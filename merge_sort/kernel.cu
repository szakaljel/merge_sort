
#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;


#define GLOBAL_ELEMENT_SIZE  1024

__device__ void merge(int * start_a,int * start_tmp,int count)
{
	int first_x;
	int second_x;
	int *current_first = start_a;
	int *last_first = current_first + (count / 2);
	int *current_second = start_a + (count / 2);
	int *last_second = current_second + (count / 2);



	for (int i = 0; i < count; i++)
	{
		second_x = current_second < last_second ? current_second[0] : 0x7FFFFFFF;
		first_x = current_first < last_first ? current_first[0] : 0x7FFFFFFF;

		start_tmp[i] = second_x < first_x ? second_x : first_x;
		current_second = second_x < first_x ? current_second+1 : current_second;
		current_first = second_x < first_x ? current_first : current_first+1;

	}

	for (int i = 0; i < count; i++)
	{
		start_a[i] = start_tmp[i];
	}

	return;
}

__global__ void kernel(int * da)
{
	__shared__ int tmp_memory[GLOBAL_ELEMENT_SIZE];
	__shared__ int swap_memory[GLOBAL_ELEMENT_SIZE];
	
	int tid = threadIdx.x;
	int activeThreads = blockDim.x;
	int jump = 2;
	int *start_a;
	int *start_tmp;

	if (tid == 0)
	{
		for (int i = 0; i < GLOBAL_ELEMENT_SIZE; i++)
		{
			tmp_memory[i] = da[i];
		}
	}
	__syncthreads();
	while (jump <= 2*blockDim.x)
	{
		if (tid < activeThreads)
		{
			start_a = tmp_memory + jump*tid;
			start_tmp = swap_memory + jump*tid;
			merge(start_a, start_tmp, jump);
			__syncthreads();
			
		}
		activeThreads = activeThreads/2;
		jump = jump*2;

	}
	if (tid == 0)
	{
		for (int i = 0; i < GLOBAL_ELEMENT_SIZE; i++)
		{
			da[i] = tmp_memory[i];
		}
	}

}


int compare(const void * a, const void * b)
{
	return (*(int*)a - *(int*)b);
}

int main()
{

	int ElementCount = GLOBAL_ELEMENT_SIZE;
	int TotalSize = ElementCount * sizeof(int);
	int *table;

	hipError_t error;
	hipSetDevice(0);
	hipSetDevice(hipDeviceMapHost);

	error = hipHostAlloc(&table, TotalSize, hipHostMallocMapped);
	
	srand(time(0));

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 1000;
	}
	

	int * da; 
	error = hipHostGetDevicePointer(&da, table, 0);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernel<<<1, ElementCount / 2 ,TotalSize*2>>> (da);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float duration;

	hipEventElapsedTime(&duration, start, stop);

	cout << duration << " ms" << endl;

	for (int i = 0; i < ElementCount; i++)
	{
		if (i % 8 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";
		
	}
	
	hipFree(da);
	hipHostFree(table);
	

	getchar();

	/*table = new int[ElementCount];

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 1000;
	}


	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	qsort(table, ElementCount, sizeof(int), compare);
	high_resolution_clock::time_point t2 = high_resolution_clock::now();

	auto duration2 = duration_cast<milliseconds>(t2 - t1).count();

	cout << duration2 <<" ms"<<endl;
	*/
	/*for (int i = 0; i < ElementCount; i++)
	{
		if (i % 8 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";

	}*/
	

/*	delete[] table;

	getchar();
	*/
    return 0;
}
