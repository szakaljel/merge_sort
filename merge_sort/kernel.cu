#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;


#define CHUNK_ELEMENT_COUNT  2048
#define FULL_ELEMENT_COUNT 128 * CHUNK_ELEMENT_COUNT

__device__ void merge(int * start_a,int * start_tmp,int count)
{

	int *current_first = start_a;
	int *last_first = current_first + (count / 2);
	int *current_second = start_a + (count / 2);
	int *last_second = current_second + (count / 2);
	int tmp;

	for (int i = 0; i < count; i++)
	{
		if (current_first < last_first)
		{
			if (current_second < last_second)
			{
				if (current_first[0] < current_second[0])
				{
					tmp = current_first[0];
					start_tmp[i] = tmp;
					current_first += 1;
				}
				else
				{
					tmp = current_second[0];
					start_tmp[i] = tmp;
					current_second += 1;
				}
			}
			else
			{
				tmp = current_first[0];
				start_tmp[i] = tmp;
				current_first += 1;
			}
		}
		else
		{
			tmp = current_second[0];
			start_tmp[i] = tmp;
			current_second += 1;
		}
	}

	for (int i = 0; i < count; i++)
	{
		start_a[i] = start_tmp[i];
	}
}

__global__ void kernel(int * da)
{
	__shared__ int tmp_memory[CHUNK_ELEMENT_COUNT];
	__shared__ int swap_memory[CHUNK_ELEMENT_COUNT];
	
	int tid = threadIdx.x;
	int offset = blockIdx.x * CHUNK_ELEMENT_COUNT;
	int activeThreads = blockDim.x;
	int jump = 2;
	int *start_a;
	int *start_tmp;

	tmp_memory[2 * tid] = da[2 * tid+offset];
	tmp_memory[2 * tid + 1] = da[2 * tid + 1+offset];

	__syncthreads();
	while (jump <= 2*blockDim.x)
	{
		if (tid < activeThreads)
		{
			start_a = tmp_memory + jump*tid;
			start_tmp = swap_memory + jump*tid;
			merge(start_a, start_tmp, jump);
			
			
		}
		activeThreads = activeThreads/2;
		jump = jump*2;
		__syncthreads();

	}
	
	da[2 * tid + offset] = tmp_memory[2 * tid];
	da[2 * tid + 1 + offset] = tmp_memory[2 * tid + 1];


}

__global__ void kernel_second_merge(int * da, int* dtmp)
{
	
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int activeThreads = gridDim.x * blockDim.x;
	int jump = CHUNK_ELEMENT_COUNT*2;
	int *start_a;
	int *start_tmp;
	while (jump <= FULL_ELEMENT_COUNT)
	{
		if (tid < activeThreads)
		{
			start_a = da + jump*tid;
			start_tmp = dtmp + jump*tid;
			merge(start_a, start_tmp, jump);
			
			
		}
		activeThreads = activeThreads/2;
		jump = jump*2;
		__syncthreads();

	}

}

int compare(const void * a, const void * b)
{
	return (*(int*)a - *(int*)b);
}


bool is_sort(int* tab,int count)
{
	for(int i=0;i<count-1;i++)
	{
		if(tab[i]>tab[i+1])
		{
			cout<<i<<endl;
			return false;
		}
	}
	return true;
}


int main()
{

	int ElementCount = FULL_ELEMENT_COUNT;
	int ChunkCount = CHUNK_ELEMENT_COUNT;
	int FullSize = ElementCount * sizeof(int);
	int ChunkSize = ChunkCount * sizeof(int);
	int *table;
	int * result;

	hipError_t error;
	hipSetDevice(0);
	hipSetDevice(hipDeviceMapHost);

	error = hipHostAlloc(&table, FullSize, hipHostMallocMapped);
	error = hipHostAlloc(&result, FullSize, hipHostMallocMapped);

	srand(time(0));

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 1000000;
	}
	


	int * da; 
	error = hipHostGetDevicePointer(&da, table, 0);
	
	int * dtmp;
	error = hipHostGetDevicePointer(&dtmp, result, 0);

	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);

	//hipEventRecord(start);

	high_resolution_clock::time_point tt1 = high_resolution_clock::now();
	kernel<<<ElementCount/ChunkCount, ChunkCount / 2 ,ChunkSize*2>>> (da);
	kernel_second_merge<<<1,(ElementCount/ChunkCount)/2>>>(da,dtmp);
	
	hipDeviceSynchronize();
	high_resolution_clock::time_point tt2 = high_resolution_clock::now();

	//hipEventRecord(stop);
	//hipEventSynchronize(stop);

	//float duration;
	auto duration = duration_cast<milliseconds>(tt2 - tt1).count();

	//hipEventElapsedTime(&duration, start, stop);
	

	cout << duration << " ms" << "  sort: "<<is_sort(table,ElementCount)<< endl;

	/*for (int i = 0; i < 8096; i++)
	{
		if (i % 16 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";
		
	}*/
	
	hipFree(da);
	hipHostFree(table);

	hipFree(dtmp);
	hipHostFree(result);
	

	getchar();

	table = new int[ElementCount];

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 1000000;
	}


	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	qsort(table, ElementCount, sizeof(int), compare);
	high_resolution_clock::time_point t2 = high_resolution_clock::now();

	auto duration2 = duration_cast<milliseconds>(t2 - t1).count();

	cout << duration2 << " ms" << "  sort: "<<is_sort(table,ElementCount)<< endl;

	/*for (int i = 0; i < ElementCount; i++)
	{
		if (i % 8 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";

	}*/
	

	delete[] table;

	getchar();
	
    return 0;
}
