
#include "hip/hip_runtime.h"

#include <cstdlib>
#include <ctime>
#include <iostream>
#include <chrono>

using namespace std;
using namespace std::chrono;


#define CHUNK_ELEMENT_COUNT  1024
#define FULL_ELEMENT_COUNT 64 * CHUNK_ELEMENT_COUNT

__device__ void merge(int * start_a,int * start_tmp,int count)
{

	int *current_first = start_a;
	int *last_first = current_first + (count / 2);
	int *current_second = start_a + (count / 2);
	int *last_second = current_second + (count / 2);
	int tmp;

	for (int i = 0; i < count; i++)
	{
		if (current_first < last_first)
		{
			if (current_second < last_second)
			{
				if (current_first[0] < current_second[0])
				{
					tmp = current_first[0];
					start_tmp[i] = tmp;
					current_first += 1;
				}
				else
				{
					tmp = current_second[0];
					start_tmp[i] = tmp;
					current_second += 1;
				}
			}
			else
			{
				tmp = current_first[0];
				start_tmp[i] = tmp;
				current_first += 1;
			}
		}
		else
		{
			tmp = current_second[0];
			start_tmp[i] = tmp;
			current_second += 1;
		}
	}

	for (int i = 0; i < count; i++)
	{
		start_a[i] = start_tmp[i];
	}
}

__global__ void kernel(int * da)
{
	__shared__ int tmp_memory[CHUNK_ELEMENT_COUNT];
	__shared__ int swap_memory[CHUNK_ELEMENT_COUNT];
	
	int tid = threadIdx.x;
	int offset = blockIdx.x * CHUNK_ELEMENT_COUNT;
	int activeThreads = blockDim.x;
	int jump = 2;
	int *start_a;
	int *start_tmp;

	tmp_memory[2 * tid] = da[2 * tid+offset];
	tmp_memory[2 * tid + 1] = da[2 * tid + 1+offset];

	__syncthreads();
	while (jump <= 2*blockDim.x)
	{
		if (tid < activeThreads)
		{
			start_a = tmp_memory + jump*tid;
			start_tmp = swap_memory + jump*tid;
			merge(start_a, start_tmp, jump);
			
			
		}
		activeThreads = activeThreads/2;
		jump = jump*2;
		__syncthreads();

	}
	
	da[2 * tid + offset] = tmp_memory[2 * tid];
	da[2 * tid + 1 + offset] = tmp_memory[2 * tid + 1];


}


int compare(const void * a, const void * b)
{
	return (*(int*)a - *(int*)b);
}

int main()
{

	int ElementCount = FULL_ELEMENT_COUNT;
	int ChunkCount = CHUNK_ELEMENT_COUNT;
	int FullSize = ElementCount * sizeof(int);
	int ChunkSize = ChunkCount * sizeof(int);
	int *table;

	hipError_t error;
	hipSetDevice(0);
	hipSetDevice(hipDeviceMapHost);

	error = hipHostAlloc(&table, FullSize, hipHostMallocMapped);
	
	srand(time(0));

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 10000;
	}
	

	int * da; 
	error = hipHostGetDevicePointer(&da, table, 0);
	//error = cudaMalloc(&d_tmp, FullSize);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	kernel<<<ElementCount/ChunkCount, ChunkCount / 2 ,ChunkSize*2>>> (da);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float duration;

	hipEventElapsedTime(&duration, start, stop);

	cout << duration << " ms" << endl;

	/*for (int i = 0; i < ElementCount; i++)
	{
		if (i % 8 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";
		
	}*/
	
	hipFree(da);
	hipHostFree(table);
	

	getchar();

	/*table = new int[ElementCount];

	for (int i = 0; i < ElementCount; i++)
	{
		table[i] = rand() % 1000;
	}


	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	qsort(table, ElementCount, sizeof(int), compare);
	high_resolution_clock::time_point t2 = high_resolution_clock::now();

	auto duration2 = duration_cast<milliseconds>(t2 - t1).count();

	cout << duration2 <<" ms"<<endl;
	*/
	/*for (int i = 0; i < ElementCount; i++)
	{
		if (i % 8 == 0)
		{
			cout << endl;
		}
		cout << table[i] << "  ";

	}*/
	

/*	delete[] table;

	getchar();
	*/
    return 0;
}
